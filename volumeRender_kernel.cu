#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2012 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 *
 */

// Simple 3D volume renderer

#ifndef _VOLUMERENDER_KERNEL_CU_
#define _VOLUMERENDER_KERNEL_CU_

#include <hip/hip_runtime_api.h>
#include <hip/hip_vector_types.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <>
#include "utility.h"
#include <time.h>
#include <thrust/random/normal_distribution.h>
#include <thrust/random/linear_congruential_engine.h>
#include <thrust/random.h>
//#include "cuda_shader.h"

typedef unsigned int  uint;
typedef unsigned char uchar;

hipArray *d_volumeArray = 0;
hipArray *d_visArray = 0;
hipArray *d_normalArray = 0;
hipArray *d_transferFuncArray1;
hipArray *d_transferFuncArray2;

typedef unsigned char VolumeType;
//typedef unsigned short VolumeType;
//typedef float4 VolumeType;

//texture<VolumeType, 3, hipReadModeNormalizedFloat> tex;         // 3D texture
texture<float, 3, hipReadModeElementType> tex_volume;         // 3D texture
texture<float, 3, hipReadModeElementType> tex_vis;         // 3D texture
texture<float, 3, hipReadModeElementType> tex_normal;         // 3D texture
texture<float4, 1, hipReadModeElementType>         transferTex; // 1D transfer function texture
texture<float4, 1, hipReadModeElementType>         transferTex_color; // 1D transfer function texture
texture<float, 1, hipReadModeElementType>         transferTex_alpha; // 1D transfer function texture

typedef struct
{
    float4 m[3];
} float3x4;

__constant__ float3x4 c_invViewMatrix;  // inverse view matrix

struct Ray
{
    float3 o;   // origin
    float3 d;   // direction
};

__device__
int intersectBox(Ray r, float3 boxmin, float3 boxmax, float *tnear, float *tfar)
{
    // compute intersection of ray with all six bbox planes
    float3 invR = make_float3(1.0f) / r.d;
    float3 tbot = invR * (boxmin - r.o);
    float3 ttop = invR * (boxmax - r.o);

    // re-order intersections to find smallest and largest on each axis
    float3 tmin = fminf(ttop, tbot);
    float3 tmax = fmaxf(ttop, tbot);

    // find the largest tmin and the smallest tmax
    float largest_tmin = fmaxf(fmaxf(tmin.x, tmin.y), fmaxf(tmin.x, tmin.z));
    float smallest_tmax = fminf(fminf(tmax.x, tmax.y), fminf(tmax.x, tmax.z));

    *tnear = largest_tmin;
    *tfar = smallest_tmax;

    return smallest_tmax > largest_tmin;
}

// transform vector by matrix (no translation)
__device__
float3 mul(const float3x4 &M, const float3 &v)
{
    float3 r;
    r.x = dot(v, make_float3(M.m[0]));
    r.y = dot(v, make_float3(M.m[1]));
    r.z = dot(v, make_float3(M.m[2]));
    return r;
}

// transform vector by matrix with translation
__device__
float4 mul(const float3x4 &M, const float4 &v)
{
    float4 r;
    r.x = dot(v, M.m[0]);
    r.y = dot(v, M.m[1]);
    r.z = dot(v, M.m[2]);
    r.w = 1.0f;
    return r;
}

__device__ uint rgbaFloatToInt(float4 rgba)
{
    rgba.x = __saturatef(rgba.x);   // clamp to [0.0, 1.0]
    rgba.y = __saturatef(rgba.y);
    rgba.z = __saturatef(rgba.z);
    rgba.w = __saturatef(rgba.w);
    return (uint(rgba.w*255)<<24) | (uint(rgba.z*255)<<16) | (uint(rgba.y*255)<<8) | uint(rgba.x*255);
}

struct RenderParam{
    uint imageW;
    uint imageH;
    float density, brightness;
    float transferOffset;
    float transferScale;
    hipExtent volumeSize;
    float min_value, max_value,gridScale_X,gridScale_Y, gridScale_Z, tstep;
    int maxSteps ;
    char visibilityOn;
};
__constant__ RenderParam c_vrParam;

__global__ void
d_render(uint *d_output)
{
    const float opacityThreshold = 0.99f;
    const float3 boxMin = make_float3(-c_vrParam.gridScale_X, -c_vrParam.gridScale_Y, -c_vrParam.gridScale_Z);
    const float3 boxMax = make_float3(c_vrParam.gridScale_X, c_vrParam.gridScale_Y, c_vrParam.gridScale_Z);
	//const float3 boxMin = make_float3(-1.0f, -1.0f, -1.0f);
    //const float3 boxMax = make_float3(1.0f, 1.0f, 1.0f);

    uint x = blockIdx.x*blockDim.x + threadIdx.x;
    uint y = blockIdx.y*blockDim.y + threadIdx.y;

    //float W = (float) imageW * gridScale_X;
    //float H = (float) imageH * gridScale_Y;

    if ((x >= c_vrParam.imageW) || (y >= c_vrParam.imageH))
        return;

    // calculate eye ray in world space
    Ray eyeRay;
    float u = (x / (float) c_vrParam.imageW)*2.0f*c_vrParam.gridScale_X-c_vrParam.gridScale_X;
    float v = (y / (float) c_vrParam.imageH)*2.0f*c_vrParam.gridScale_Y-c_vrParam.gridScale_Y;
    //float u = (x / (float) W)*2.0f-1.0f;
    //float v = (y / (float) H)*2.0f-1.0f;
    //float w = (z / (float) imageH)*2.0f-1.0f;

    eyeRay.o = make_float3(mul(c_invViewMatrix, make_float4(0.0f, 0.0f, 0.0f, 1.0f)));
    eyeRay.d = normalize(make_float3(u, v, -2.0f));
    eyeRay.d = mul(c_invViewMatrix, eyeRay.d);

    // find intersection with box
    float tnear, tfar;
    {
        int hit = intersectBox(eyeRay, boxMin, boxMax, &tnear, &tfar);
        if (!hit) return;
    }

    if (tnear < 0.0f) tnear = 0.0f;     // clamp to near plane
    // march along ray from front to back, accumulating color
    float4 sum = make_float4(0.0f);
    float t = tnear;
    float3 pos = eyeRay.o + eyeRay.d*tnear;
    float3 step = eyeRay.d*c_vrParam.tstep;
    float sample;
    float value_range = c_vrParam.max_value - c_vrParam.min_value;
	float temp_range = 1.0f/value_range;

    for (int i=0; i<c_vrParam.maxSteps; i++)
    {
        // read from 3D texture
        // remap position to [0, 1] coordinates
        //sample = tex3D(tex, pos.x*0.5f+0.5f, pos.y*0.5f+0.5f, pos.z*0.5f+0.5f);
        float x = (pos.x/(2.0*c_vrParam.gridScale_X)+0.5)*(c_vrParam.volumeSize.width-1);
        float y = (pos.y/(2.0*c_vrParam.gridScale_Y)+0.5)*(c_vrParam.volumeSize.height-1);
        float z = (pos.z/(2.0*c_vrParam.gridScale_Z)+0.5)*(c_vrParam.volumeSize.depth-1);
        sample = tex3D(tex_volume, x, y, z);
		
        sample = (sample-c_vrParam.min_value)*temp_range;    // rmap scale to [0 , 1]
        sample = (sample-c_vrParam.transferOffset)*c_vrParam.transferScale;
	
        // lookup in transfer function texture
        
		//float4 col = tex1D(transferTex, (sample-transferOffset)*transferScale);

        float4 col = tex1D(transferTex_color, sample);
        float alpha = tex1D(transferTex_alpha, sample);

        if (c_vrParam.visibilityOn) {
            float visible = tex3D(tex_vis, x, y, z);
            //alpha = visible;
            if (visible == 0) {
                float m = (col.x+col.y+col.z)/3.f;  // lightness
                col.x = col.y = col.z = m;
                alpha = fmin(alpha, .005f);
            }else {
                alpha = visible;
            }
        }

        col.w = alpha;
		
        col.w *= c_vrParam.density;

        // "under" operator for back-to-front blending
        //sum = lerp(sum, col, col.w);

        // pre-multiply alpha
        col.x *= col.w;
        col.y *= col.w;
        col.z *= col.w;

        // "over" operator for front-to-back blending
        sum += col*(1.0f - sum.w);

        // exit early if opaque
        if (sum.w > opacityThreshold)
            break;

        t += c_vrParam.tstep;

        if (t > tfar) break;

        pos += step;
    }

    sum *= c_vrParam.brightness;

    // write output color
    d_output[y*c_vrParam.imageW + x] = rgbaFloatToInt(sum);

}
__global__ void
gmm_kernel(thrust::random::uniform_real_distribution<float> dist, thrust::minstd_rand rng, float *gmm_V, float* gmm_P, int xdim, int ydim, int zdim, int distib_num, unsigned int m_seed2)
{
#if 0
	uint x = blockIdx.x*blockDim.x + threadIdx.x;
    uint y = blockIdx.y*blockDim.y + threadIdx.y;
	uint z = blockIdx.y*blockDim.z + threadIdx.z;
	uint index = z*xdim*ydim + y*xdim + x;

    //float sum,uni_rand,mean,stdev;
    bool flag=false;
    int gaussian_index;
	
    sum=gmm_P[index+2]; //weighted of first gaussian
//	uni_rand = dist(rng);
/*	
	for(int j=0;j<distib_num-1;j++)
	{
		if(uni_rand<sum)
		{
			gmm_V[index] = j;
			flag = true;
			break;
		}
		else
			sum += gmm_P[index+(j+2)*3-1];
	}
	
	if(!flag)
		gmm_V[index] = distib_num-1;

	mean = gmm_P[(int)(index+gmm_V[index]*3)];
	stdev = gmm_P[(int)(index+gmm_V[index]*3+1)];
	
	thrust::minstd_rand rng2;
	rng2.seed(m_seed2);
	thrust::random::experimental::normal_distribution<float> dist2(mean,stdev);
	gmm_V[index] = dist2(rng2);
	*/
#endif
}

#if 0
__global__ void
Fuzzy_IsoSusrface(uint *v_output, uint xdim, uint ydim, uint zdim, float iso, int block_size, int binWidth,float minvalue)
{
	int index;
	int above,under;
	int offset = block_size*0.5;
	int iso_location = ((iso-minvalue)/binWidth)+0.5;
	int iter =0;

	float *block_data;
	float *hist;

	float score;

	uint x = blockIdx.x*blockDim.x + threadIdx.x;
    uint y = blockIdx.y*blockDim.y + threadIdx.y;
	uint z = blockIdx.z*blockDim.z + threadIdx.z;

	if(x>xdim || y>ydim || z>zdim)
		return;

	under = 0;
	above = 0;

	////////////////////////////////////////////////
	int z_lowerbond = (z-offset)>=0 ? z-offset:0;
	int y_lowerbond = (y-offset)>=0 ? y-offset:0;
	int x_lowerbond = (x-offset)>=0 ? x-offset:0;

	int z_upperbond = (z+offset)<zdim ? z+offset:zdim-1;
	int y_upperbond = (y+offset)<ydim ? y+offset:ydim-1;
	int x_upperbond = (x+offset)<xdim ? x+offset:xdim-1;
	
	hipExtent size = make_hipExtent(x_upperbond-x_lowerbond+1, y_upperbond-y_lowerbond+1, z_upperbond-z_lowerbond+1);
	hipArray *d_blockArray = 0;
	hipChannelFormatDesc channelDesc = hipCreateChannelDesc<float>();
    checkCudaErrors(hipMalloc3DArray(&d_blockArray, &channelDesc, size));

	for(int z=z_lowerbond;z<=z_upperbond;z++)
		for(int y=y_lowerbond;y<=y_upperbond;y++)
			for(int x=x_lowerbond;x<=x_upperbond;x++)
			{
				int idx = x + y*xdim + z*xdim*ydim;
				d_blockArray[iter++] = d_volumeArray[idx];							
			}

////////////////////////////////////////////////////////////
				block_data = &tempdata[0];

				hist = info->histogram(block_data,tempdata.size(),binWidth,minvalue,maxvalue);

				for(int iter=0;iter<iso_location;iter++)
				{
					if(hist[iter]>0)
						under++;
				}

				for(int iter=iso_location;iter<info->hist_gridsize;iter++)
				{
					if(hist[iter]>0)
						above++;
				}
				if(under==0)
					score = above;
				else if(above==0)
					score = under;
				else
					score = (float)above/(float)under - (float)under/(float)above;
				/*
				if(under==0 || above==0)
					score = 1;
				else					
					score = above>=under ? (1-(float)under/(float)above):(1-(float)above/(float)under);
					*/
				V[index] = (score);	

	checkCudaErrors(hipFreeArray(d_blockArray));
}
#endif

extern "C"
void setTextureFilterMode(bool bLinearFilter)
{
    tex_volume.filterMode = bLinearFilter ? hipFilterModeLinear : hipFilterModePoint;
	tex_normal.filterMode = bLinearFilter ? hipFilterModeLinear : hipFilterModePoint;
}

extern "C"
void init_volume_Cuda(float *h_volume, hipExtent volumeSize)
{
    // create 3D array
    hipChannelFormatDesc channelDesc = hipCreateChannelDesc<float>();
    checkCudaErrors(hipMalloc3DArray(&d_volumeArray, &channelDesc, volumeSize));

    // copy data to 3D array
    hipMemcpy3DParms copyParams = {0};
    copyParams.srcPtr   = make_hipPitchedPtr(h_volume, volumeSize.width*sizeof(float), volumeSize.width, volumeSize.height);
    copyParams.dstArray = d_volumeArray;
    copyParams.extent   = volumeSize;
    copyParams.kind     = hipMemcpyHostToDevice;
    checkCudaErrors(hipMemcpy3D(&copyParams));

    // set texture parameters
    tex_volume.normalized = false;                      // access with normalized texture coordinates
    tex_volume.filterMode = hipFilterModeLinear;      // linear interpolation
	//tex.filterMode = hipFilterModePoint;
	tex_volume.channelDesc = channelDesc;
    tex_volume.addressMode[0] = hipAddressModeClamp;  // clamp texture coordinates
    tex_volume.addressMode[1] = hipAddressModeClamp;
	tex_volume.addressMode[2] = hipAddressModeClamp;

    // bind array to 3D texture
    checkCudaErrors(hipBindTextureToArray(tex_volume, d_volumeArray, channelDesc));

}

extern "C"
void init_visibility_Cuda(float *h_volume, hipExtent volumeSize)
{
    // create 3D array
    hipChannelFormatDesc channelDesc = hipCreateChannelDesc<float>();
    checkCudaErrors(hipMalloc3DArray(&d_visArray, &channelDesc, volumeSize));

    // copy data to 3D array
    hipMemcpy3DParms copyParams = {0};
    copyParams.srcPtr   = make_hipPitchedPtr(h_volume, volumeSize.width*sizeof(float), volumeSize.width, volumeSize.height);
    copyParams.dstArray = d_visArray;
    copyParams.extent   = volumeSize;
    copyParams.kind     = hipMemcpyHostToDevice;
    checkCudaErrors(hipMemcpy3D(&copyParams));

    // set texture parameters
    tex_vis.normalized = false;                      // access with normalized texture coordinates
    tex_vis.filterMode = hipFilterModeLinear;      // linear interpolation
    //tex.filterMode = hipFilterModePoint;
    tex_vis.channelDesc = channelDesc;
    tex_vis.addressMode[0] = hipAddressModeClamp;  // clamp texture coordinates
    tex_vis.addressMode[1] = hipAddressModeClamp;
    tex_vis.addressMode[2] = hipAddressModeClamp;

    // bind array to 3D texture
    checkCudaErrors(hipBindTextureToArray(tex_vis, d_visArray, channelDesc));

}

extern "C"
void init_normal_Cuda(myvector4 *h_volume, hipExtent volumeSize)
{
    // create 3D array
    hipChannelFormatDesc channelDesc = hipCreateChannelDesc<float4>();
    checkCudaErrors(hipMalloc3DArray(&d_normalArray, &channelDesc, volumeSize,1));

    // copy data to 3D array
    hipMemcpy3DParms copyParams = {0};
    copyParams.srcPtr   = make_hipPitchedPtr(h_volume, volumeSize.width*4*sizeof(float), volumeSize.width, volumeSize.height);
    copyParams.dstArray = d_normalArray;
    copyParams.extent   = volumeSize;
    copyParams.kind     = hipMemcpyHostToDevice;
    checkCudaErrors(hipMemcpy3D(&copyParams));

    // set texture parameters
    tex_normal.normalized = false;                      // access with normalized texture coordinates
    tex_normal.filterMode = hipFilterModeLinear;      // linear interpolation
	//tex.filterMode = hipFilterModePoint;
	tex_normal.channelDesc = channelDesc;
    tex_normal.addressMode[0] = hipAddressModeClamp;  // clamp texture coordinates
    tex_normal.addressMode[1] = hipAddressModeClamp;
	tex_normal.addressMode[2] = hipAddressModeClamp;

    // bind array to 3D texture
    checkCudaErrors(hipBindTextureToArray(tex_normal, d_normalArray, channelDesc));

}

extern "C"
void TransferFunc(float *transf_color, int num_color, float *transf_alpha, int num_alpha)
{
	
	float4 *t_color = new float4[num_color];
	for(int i=0;i<num_color;i++)
	{
		t_color[i].x = transf_color[i*4];
		t_color[i].y = transf_color[i*4+1];
		t_color[i].z = transf_color[i*4+2];
		t_color[i].w = transf_color[i*4+3];
	}
	
    hipChannelFormatDesc channelDesc2 = hipCreateChannelDesc<float4>();
	
    checkCudaErrors(hipMallocArray(&d_transferFuncArray1, &channelDesc2, num_color, 1));
	checkCudaErrors(hipMemcpyToArray(d_transferFuncArray1, 0, 0, t_color, num_color*4*sizeof(float), hipMemcpyHostToDevice));

	transferTex_color.filterMode = hipFilterModeLinear;
    transferTex_color.normalized = true;    // access with normalized texture coordinates
    transferTex_color.addressMode[0] = hipAddressModeClamp;   // wrap texture coordinates

    // Bind the array to the texture
    checkCudaErrors(hipBindTextureToArray(transferTex_color, d_transferFuncArray1, channelDesc2));


	hipChannelFormatDesc channelDesc3 = hipCreateChannelDesc<float>();
	
    checkCudaErrors(hipMallocArray(&d_transferFuncArray2, &channelDesc3, num_alpha, 1));
	checkCudaErrors(hipMemcpyToArray(d_transferFuncArray2, 0, 0, transf_alpha, num_alpha*sizeof(float), hipMemcpyHostToDevice));

	transferTex_alpha.filterMode = hipFilterModeLinear;
    transferTex_alpha.normalized = true;    // access with normalized texture coordinates
    transferTex_alpha.addressMode[0] = hipAddressModeClamp;   // wrap texture coordinates

    // Bind the array to the texture
    checkCudaErrors(hipBindTextureToArray(transferTex_alpha, d_transferFuncArray2, channelDesc3));
	delete [] t_color;
}


extern "C"
void freeCudaVolumeBuffers()
{
    checkCudaErrors(hipFreeArray(d_volumeArray));
	checkCudaErrors(hipFreeArray(d_normalArray));
    checkCudaErrors(hipFreeArray(d_visArray));

}

extern "C"
void freeCudaTransferFuncBuffers()
{    
    checkCudaErrors(hipFreeArray(d_transferFuncArray1));
	checkCudaErrors(hipFreeArray(d_transferFuncArray2));
}


extern "C"
void render_kernel(dim3 gridSize, dim3 blockSize, uint *d_output, uint imageW, uint imageH,
                   float density, float brightness, float transferOffset, float transferScale, 
				   hipExtent volumeSize, float min_value, float max_value, float gridScale_X, float gridScale_Y, float gridScale_Z, float tstep)
				   
{
    RenderParam param;
    param.imageW = imageW;
    param.imageH = imageH;
    param.density= density;
    param.brightness = brightness;
    param.transferOffset= transferOffset;
    param.transferScale= transferScale;
    param.volumeSize=volumeSize;
    param.min_value=min_value;
    param.max_value= max_value;
    param.gridScale_X=gridScale_X;
    param.gridScale_Y=gridScale_Y;
    param.gridScale_Z=gridScale_Z;
    param.tstep=tstep;
    param.maxSteps = 1000;
    param.visibilityOn = true; // set on for now
    hipMemcpyToSymbol(HIP_SYMBOL(c_vrParam), &param, sizeof(RenderParam));
    d_render<<<gridSize, blockSize>>>(d_output);
}

extern "C"
void copyInvViewMatrix(float *invViewMatrix, size_t sizeofMatrix)
{
    checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(c_invViewMatrix), invViewMatrix, sizeofMatrix));
}


extern "C"
void gmm_compute_kernel(dim3 gridSize, dim3 blockSize,int size, float *gmm_V,float *gmm_P, hipExtent volumeSize, int xdim, int ydim, int zdim, int distib_num)
{

	unsigned int m_seed = rand();
	thrust::minstd_rand rng(size);
	rng.seed(m_seed);
	thrust::random::uniform_real_distribution<float> dist(0,1);
	unsigned int m_seed2 = rand();

	gmm_kernel<<<gridSize, blockSize>>>(dist,rng,gmm_V,gmm_P,xdim,ydim,zdim,distib_num,m_seed2);


#if 0
	float sum,uni_rand;
	bool flag;
	for(int i=0;i<size;i++)
	{
			sum=gmm_P[i].parameter[2]; //weighted of first gaussian 
		    flag=false;
			rng.discard(1);
			uni_rand = dist(rng);
			for(int j=0;j<gmm_P[i].mixtured_numer-1;j++)
			{
				if(uni_rand<sum)
				{
					gmm_V[i] = j;
					flag = true;
					break;
				}
				else
					sum += gmm_P[i].parameter[(j+2)*3-1];
			}
			if(!flag)
				gmm_V[i] = gmm_P[i].mixtured_numer-1;

			//g.data.volume[i] = CV->generate_gaussian_random(g.data.gmm[i].parameter[gmm_index[i]*3],g.data.gmm[i].parameter[gmm_index[i]*3+1]);
			
			unsigned int m_seed2 = rand();
			thrust::minstd_rand rng2;
			rng2.seed(m_seed2);
			thrust::random::experimental::normal_distribution<float> dist2(gmm_P[i].parameter[gmm_V[i]*3],gmm_P[i].parameter[gmm_V[i]*3+1]);
			gmm_V[i] = dist2(rng2);
	}

// copy data to 3D array
	hipChannelFormatDesc channelDesc = hipCreateChannelDesc<float>();
    hipMemcpy3DParms copyParams = {0};
    copyParams.srcPtr   = make_hipPitchedPtr(gmm_V, volumeSize.width*sizeof(float), volumeSize.width, volumeSize.height);
    copyParams.dstArray = d_volumeArray;
    copyParams.extent   = volumeSize;
    copyParams.kind     = hipMemcpyHostToDevice;
    checkCudaErrors(hipMemcpy3D(&copyParams));

    // set texture parameters
    tex_volume.normalized = false;                      // access with normalized texture coordinates
    tex_volume.filterMode = hipFilterModeLinear;      // linear interpolation
	//tex.filterMode = hipFilterModePoint;
	tex_volume.channelDesc = channelDesc;
    tex_volume.addressMode[0] = hipAddressModeClamp;  // clamp texture coordinates
    tex_volume.addressMode[1] = hipAddressModeClamp;
	tex_volume.addressMode[2] = hipAddressModeClamp;
	checkCudaErrors(hipBindTextureToArray(tex_volume, d_volumeArray, channelDesc));
#endif
}
#endif // #ifndef _VOLUMERENDER_KERNEL_CU_
